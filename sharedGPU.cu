/*
 Name: Ehsan Foroumandi
*/



#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <sys/time.h>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;
double gettime(void) {
		struct timeval tval;

		  gettimeofday(&tval, NULL);

  	return( (double)tval.tv_sec + (double)tval.tv_usec/1000000.0 );
}

__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}



int main() {
 
  int N = 10000;
  double t1, t2;
	t1 = gettime();

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 20;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

 

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  t2 = gettime();
	printf("Run time for the shared: %f\n", t2-t1);

  return 0;
}
